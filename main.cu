#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdint.h>
#include <iostream>

// On importe les fichiers pour utiliser un floutage gaussien
#define STB_IMAGE_IMPLEMENTATION
#include "./stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "./stb_image_write.h"

#define PI 3.14159265358979323846

/*Fonction executable sur GPU
    Permet de calculer la valeur du noyau gaussien,
    plus la valeur de l'entrée `sigma` sera élevé et plus l'image sera flouter
    plus cette valeur est élevée, plus le temps de calcul sera long
*/
__device__ float gaussian(float x, float sigma)
{
    return (1.0 / (2 * PI * sigma * sigma)) * exp(-(x * x) / (2 * sigma * sigma));
}

/* Fonction de calcul des nouvelles valeurs des pixel en appliquant le filtre gaussien
    Paramètres : paramètres de l'image (largeur, hauteur, nombre de canaux de couleur) + sigma pour l'écart type du noyau gaussien
    Les threads calculent la valeur d'un pixel de l'image de sortie après l'application du filtre gaussien.
*/
__global__ void applyGaussianBlur(const uint8_t *inputPixels, uint8_t *outputPixels, int width, int height, int channels, float sigma)
{
    int radius = (int)(sigma * 3);
    int size = 2 * radius + 1;

    float kernel[61];

    // construction du noyau de convo gaussien
    float sum = 0;
    for (int i = 0; i < size; i++)
    {
        kernel[i] = gaussian(i - radius, sigma);
        sum += kernel[i];
    }

    // Nomalisation du noyau
    /*Le masque resemble à :
        { kernel[0] / sum, kernel[0] / sum, kernel[0] / sum },
        { kernel[1] / sum, kernel[1] / sum, kernel[1] / sum },
        { kernel[2] / sum, kernel[2] / sum, kernel[2] / sum }
    */
    for (int i = 0; i < size; i++)
    {
        kernel[i] /= sum;
    }

    // x et y => pixel de l'image
    // associe à chaque thread un pixel donnée
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) // vérifie que les variables x et y ne depasse pas les limites de la taille de l'image
    {
        // Appliquer le noyau de convolution on commence par passer sur chacun des canaux sachant que chaque canal est un tableau de pixels car c est une image RGB
        for (int c = 0; c < channels; c++)
        {
            // Initialiser la nouvelle valeur du pixel
            float newValue = 0.0;
            for (int ky = -radius; ky <= radius; ky++)
            {
                for (int kx = -radius; kx <= radius; kx++)
                {
                    int px = x + kx;
                    int py = y + ky;

                    if (px >= 0 && px < width && py >= 0 && py < height)
                    {
                        newValue += inputPixels[(py * width + px) * channels + c] * kernel[kx + radius] * kernel[ky + radius];
                    }
                }
            }
            outputPixels[(y * width + x) * channels + c] = (uint8_t)newValue;
        }
    }
}

int main()
{
    /*Chargement de l'image ("input.bmp") avec allocation mémoire CPU
        +stocke ses pixels dans un tab : "inputPixels"
        +déclaration du tab des nouvelles valeur après filtre : "outpuPixels"
    */
    int width, height, channels;
    uint8_t *inputPixels = stbi_load("./input.bmp", &width, &height, &channels, 0);
    if (!inputPixels)
    {
        printf("Impossible de charger l'image.\n");
        return 1;
    }

    uint8_t *outputPixels = (uint8_t *)malloc(width * height * channels * sizeof(uint8_t));
    if (!outputPixels)
    {
        printf("Erreur lors de l'allocation de mémoire.\n");
        stbi_image_free(inputPixels);
        return 1;
    }

    float sigma = 10.0; // L'écart type du noyau gaussien

    uint8_t *d_inputPixels, *d_outputPixels; // init des var alloué sur cuda pour calculs des pixels; variable d'entrée + variable de sortie
    hipError_t cudaStatus;

    /*Allocation mémoire sur le GPU*/
    cudaStatus = hipMalloc((void **)&d_inputPixels, width * height * channels * sizeof(uint8_t)); // tab d_inputPixels => pixels de l'image d'entrée (dans cuda)
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }
    cudaStatus = hipMalloc((void **)&d_outputPixels, width * height * channels * sizeof(uint8_t)); // tab d_outputPixels => pixels de l'image en sortie (de cuda)
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_inputPixels);
        return 1;
    }

    /*transfert data CPU vers GPU*/
    cudaStatus = hipMemcpy(d_inputPixels, inputPixels, width * height * channels * sizeof(uint8_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_inputPixels);
        hipFree(d_outputPixels);
        return 1;
    }

    /*Calcul de la configuration des blocs et des grilles*/
    dim3 threadsPerBlock(16, 16);                                                                                              // nb de thread par bloc
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y); // taille des grilles

    // Appliquer le flou gaussien (appel du kernel cuda)
    applyGaussianBlur<<<numBlocks, threadsPerBlock>>>(d_inputPixels, d_outputPixels, width, height, channels, sigma);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_inputPixels);
        hipFree(d_outputPixels);
        return 1;
    }

    // Copie du résultat du GPU vers le CPU
    hipMemcpy(outputPixels, d_outputPixels, width * height * channels * sizeof(uint8_t), hipMemcpyDeviceToHost);

    // enregistrement de l'image floutée
    stbi_write_bmp("output.bmp", width, height, channels, outputPixels);

    // Libérer la mémoire alloué (CPU et GPU)
    stbi_image_free(inputPixels);
    free(outputPixels);
    hipFree(d_inputPixels);
    hipFree(d_outputPixels);

    printf("Image floutée enregistrée sous : output.bmp\n");

    return 0;
}